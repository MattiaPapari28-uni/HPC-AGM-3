#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "covariance.h"
#define BLOCK_DIM 32


__global__ void kernel_covariance(DATA_TYPE float_n, DATA_TYPE* __restrict__ data, DATA_TYPE* __restrict__ mean, DATA_TYPE* __restrict__ symmat)
{   
    // Compute the row (i) and column (j) indices for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y;
    int index = j * blockDim.x * gridDim.x + i; //indice globale
    // Ensure we are within bounds
    if (index < M * N and tid < M) {
        data[index] = ((DATA_TYPE)i * j) / M;
        symmat[index] = 0.0;
        // valutare utilizzo __device__ e inline, per richiamre funzioni con parti di codice pulizia codice

        /* Determine mean of column vectors of input data matrix */
        mean[tid] = 0.0;
        __syncthreads();
        
        atomicAdd(mean[tid], data[index]);
        __syncthreads();

        if (atomicCAS(&flags[tid], 0, 1) == 0) {
            // Esegui la divisione una sola volta
            mean[tid] /= float_n;
        }        

        /*
        for (j < _PB_M; j++)
          if(threadIdx.x == j)
              lock.lock();
              mean[j] /= float_n;
              lock.unlock();
        */
        __syncthreads();
       
        atomicSub(data[index], mean[tid]);
        //__syncthreads();

        __shared__ DATA_TYPE temp = 0.0;

        if(tid+1 < M)
          temp += data[i][tid] * data[i][tid+1];


    }

  /*int i, j, j1, j2;

  for (i = 0; i < M; i++)
    for (j = 0; j < N; j++)
      data[i j] = ((DATA_TYPE) i*j) / M;

  
    for (j = 0; j < M; j++)
      {
        mean[j] = 0.0;
	for (i = 0; i < N; i++)
	  mean[j] += data[i j];
	mean[j] /= float_n;
      }
      
    
    for (i = 0; i < N; i++)
      for (j = 0; j < M; j++)
	      data[i j] -= mean[j];
      
    
    /*for (j1 = 0; j1 < M; j1++)
        for (j2 = j1; j2 < M; j2++)
	      {
          symmat[j1 j2] = 0.0;
	        for (i = 0; i < N; i++)
	          symmat[j1][j2] += data[i][j1] * data[i][j2];
	        symmat[j2][j1] = symmat[j1][j2];
        }*/
}

int main(int argc, char** argv)
{
  /* Retrieve problem size. */

  double wt;
  struct timespec rt[2];
  /* Variable declaration/allocation. */
  DATA_TYPE float_n = 1.2;
  DATA_TYPE *h_symmat;
  DATA_TYPE *d_mean, *d_data, *d_symmat;
  int* d_flags;

  hipMalloc(&d_flags, size * sizeof(int));
  hipMemset(d_flags, 0, size * sizeof(int));
  hipHostMalloc((void**)&h_symmat,sizeof(DATA_TYPE) * M * M); 
  hipMalloc((void**)&d_data, sizeof(DATA_TYPE) * M * N);
  hipMalloc((void**)&d_mean, sizeof(DATA_TYPE) * M);
  hipMalloc((void**)&d_symmat, sizeof(DATA_TYPE) * M * M);
  
  /* Start timer. */
  clock_gettime(CLOCK_REALTIME, rt + 0); // non va dopo?
  dim3 dimBlock(BLOCK_DIM, BLOCK_DIM); 
  dim3 dimGrid(((N+BLOCK_DIM-1)/BLOCK_DIM)/2, ((N+BLOCK_DIM-1)/BLOCK_DIM)/2);
  /* Run kernel. */
  kernel_covariance<<<dimGrid,dimBlock>>>(float_n, d_data, d_mean, d_symmat);  
  hipMemcpy(h_symmat, d_symmat, sizeof(DATA_TYPE) * M * M, hipMemcpyDeviceToHost);
  /* Stop and print timer. */
  clock_gettime(CLOCK_REALTIME, rt + 1);
  wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
  printf("GEMM (Host) : %9.3f sec %9.1f GFLOPS\n", wt, 2.0 * N * N * N / (1.0e9 * wt));


  /* Be clean. */
  hipFree(d_data);
  hipFree(d_symmat);
  hipFree(d_mean);
  hipHostFree(h_symmat);

  return 0;
}